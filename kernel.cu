#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>

using namespace std;

// Taille de la matrice (modifiable)
#define N 4

// Kernel de transposition avec mémoire partagée et padding
__global__ void matrixTransposeNoPadding(float* input, float* floatoutput) {
    __shared__ float tile[32][32]; // Shared memory without padding

    int x = blockIdx.x * 32 + threadIdx.x; // Global index x
    int y = blockIdx.y * 32 + threadIdx.y; // Global index y

    // Load data into shared memory
    if (x < N && y < N) {
        tile[threadIdx.y][threadIdx.x] = input[y * N + x];
    }

    __syncthreads(); // Synchronize threads to ensure data is loaded

    // Transpose indices for output
    x = blockIdx.y * 32 + threadIdx.x;
    y = blockIdx.x * 32 + threadIdx.y;

    // Write transposed data to global memory
    if (x < N && y < N) {
        floatoutput[y * N + x] = tile[threadIdx.x][threadIdx.y];
    }
}

// Fonction principale
int main() {
    // Allocation de la mémoire hôte
    size_t size = N * N * sizeof(float);
    float* h_input = (float*)malloc(size);
    float* h_output = (float*)malloc(size);

    // Initialisation de la matrice d'entrée
    cout << "Initial: " << endl;
    for (int i = 0; i < N * N; ++i) {
        h_input[i] = static_cast<float>(i);
        cout << h_input[i] << endl;
    }

    // Allocation de la mémoire device
    float* d_input, * d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Copie de la matrice d'entrée vers le device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Dimensions de la grille et des blocs
    dim3 block(32, 32); // Taille du bloc
    dim3 grid((N + 31) / 32, (N + 31) / 32); // Taille de la grille

    // Exécution du kernel
    matrixTransposeShared << <grid, block >> > (d_input, d_output);
    hipDeviceSynchronize();

    // Copie des résultats vers l'hôte
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    // Vérification du résultat
    bool success = true;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            if (h_output[j * N + i] != h_input[i * N + j]) {
                success = false;
                break;
            }
        }
    }

    if (success) {
        cout << "Transposition correcte !" << endl;

        cout << "Final: " << endl;
        for (int i = 0; i < N * N; ++i) {
            cout << h_output[i] << endl;
        }
    }
    else {
        cout << "Erreur dans la transposition." << endl;
    }

    // Libération de la mémoire
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}
